/*
 * Jonathan Doman
 * jonathan.doman@gmail.com
 */

#include "Util.h"

#include <iostream>

using namespace std;

bool errorOccurred( hipError_t err, const char* operation )
{
   if( err != hipSuccess )
   {
      cout << "Error during " << operation << ": " << err << endl;
      return true;
   }
   return false;
}
