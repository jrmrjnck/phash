
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void kernel( int* a )
{
   int tid = threadIdx.x;
   a[tid] *= 2;
}

bool errorOccurred( hipError_t err, std::string operation )
{
   if( err != hipSuccess )
   {
      cout << "Error during " << operation << ": " << err << endl;
      return true;
   }
   return false;
}

#define CHECK_ERROR(err,op) do{if(errorOccurred(err,op)) return;}while(false)

void kernelWrapper( int* a, int N )
{
   int* deviceKeys;
   size_t size = N * sizeof(int);
   hipError_t err = hipMalloc( &deviceKeys, size );
   CHECK_ERROR(err,"malloc");
   err = hipMemcpy( deviceKeys, a, size, hipMemcpyHostToDevice );
   CHECK_ERROR(err,"memcpy to");

   kernel<<<1,N>>>( deviceKeys );

   err = hipMemcpy( a, deviceKeys, size, hipMemcpyDeviceToHost );
   CHECK_ERROR(err,"memcpy from");
   err = hipFree( deviceKeys );
   CHECK_ERROR(err,"free");
}
