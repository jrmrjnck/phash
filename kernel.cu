#include "hip/hip_runtime.h"
/*
 * Jonathan Doman
 * jonathan.doman@gmail.com
 */

#include "kernel.h"

#include <iostream>
#include <cassert>
#include <cstdio>

using namespace std;

namespace
{
const float LOAD_FACTOR = 0.8;
const Key NULL_KEY = 0;
const Slot NULL_SLOT = 0;

// Device side key-value data
Key*      _keys;
Value*    _values;
uint32_t* _params;

Slot* _table;

int  _inputSize;
dim3 _grid;
dim3 _block;

struct TableState
{
   Slot* table;
   int capacity;
   uint32_t* params;
   int paramIdx;
   int maxProbes;
};

TableState* _tableState;

// Slot utility functions
__device__
Slot makeSlot( Key k, Value v )
{
   return (static_cast<Slot>(k) << CHAR_BIT*sizeof(v)) | v;
}
__device__
Key slotKey( Slot s )
{
   return s >> CHAR_BIT*sizeof(Value);
}
/*__device__*/
/*Value slotValue( Slot s )*/
/*{*/
   /*return s & ((1ULL << CHAR_BIT*sizeof(Value))-1);*/
/*}*/

// Initialize table
__global__
void initTable( bool cuckoo, TableState* ts, int capacity, Slot* tableSlots, uint32_t* params, int numParams, int inputSize )
{
   ts->table    = tableSlots;
   ts->capacity = capacity;
   ts->params   = params;
   ts->paramIdx = numParams - 1;
   
   // Max probe heuristics from Alcantara
   if( cuckoo )
      ts->maxProbes = 7 * log2f( inputSize );
   else
      ts->maxProbes = min( inputSize, 10000 );

   printf( "maxProbes = %d\n", ts->maxProbes );
}

__device__
uint32_t hash( const TableState* ts, Key key, int offset = 0 )
{
   offset *= 2;
   uint64_t a = ts->params[ts->paramIdx-offset];
   uint64_t b = ts->params[ts->paramIdx-offset-1];
   return (a*key + b) % 4294967291U;
}

// Kernel to insert items using quadratic probing
__global__
void quadInsert( TableState* ts, Key* keys, Value* values )
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   Slot newEntry = makeSlot( keys[tid], values[tid] );
   uint32_t index = hash( ts, keys[tid] ) % ts->capacity;

   for( int i = 1; i <= ts->maxProbes; ++i )
   {
      // nvcc requires these ridiculous casts even though the types are the same
      Slot oldEntry = atomicCAS( reinterpret_cast<unsigned long long*>(ts->table+index),
                                 static_cast<unsigned long long>(NULL_SLOT),
                                 static_cast<unsigned long long>(newEntry) );

      // The swap was successful
      if( oldEntry == NULL_SLOT )
         return;

      // The swap was unsuccessful
      index = (index + i*i) % ts->capacity;
   }

   // Couldn't find a spot - rehash
   printf( "Insert (%u,%u) failed\n", keys[tid], values[tid] );
   // TODO: rehash - I've never actually seen a failed insertion
}

// Kernel to query an item using quadratic probing
__global__
void quadQuery( TableState* ts, Key* keys, Value* values )
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   Key key = keys[tid];
   uint32_t index = hash( ts, key ) % ts->capacity;

   for( int i = 1; i <= ts->maxProbes; ++i )
   {
      Slot entry = ts->table[index];
      Key k = slotKey( entry );

      if( k == key )
         return;
      if( k == NULL_KEY )
         break;

      index = (index + i*i) % ts->capacity;
   }

   // Should never fail except for invalid keys
   printf( "Query for %u failed\n", key );
}

__global__
void cuckooInsert( TableState* ts, Key* keys, Value* values )
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   Key key = keys[tid];
   Value value = values[tid];
   Slot entry = makeSlot( key, value );

   uint32_t idx[5];
   idx[0] = hash( ts, key, 0 ) % ts->capacity;

   for( int i = 0; i <= ts->maxProbes; ++i )
   {
      // Exchange items
      entry = atomicExch( reinterpret_cast<unsigned long long*>(&ts->table[idx[0]]), entry );

      key = slotKey( entry );

      // If the displaced item is null, we're done
      if( key == NULL_KEY )
         return;

      // Otherwise find a new location for the displaced item
      idx[1] = hash( ts, key, 0 ) % ts->capacity;
      idx[2] = hash( ts, key, 1 ) % ts->capacity;
      idx[3] = hash( ts, key, 2 ) % ts->capacity;
      idx[4] = hash( ts, key, 3 ) % ts->capacity;

           if( idx[0] == idx[1] ) idx[0] = idx[2];
      else if( idx[0] == idx[2] ) idx[0] = idx[3];
      else if( idx[0] == idx[3] ) idx[0] = idx[4];
      else                        idx[0] = idx[1];
   }

   printf( "tid %d: Insert (%u,%u) failed\n", tid, key, value );
   // TODO: rehash, but I've never actually seen a failed insertion
}

__global__
void cuckooQuery( TableState* ts, Key* keys, Value* values )
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   Key key = keys[tid];

   // Compute all possible locations
   uint32_t idx[4];
   idx[0] = hash( ts, key, 0 );
   idx[1] = hash( ts, key, 1 );
   idx[2] = hash( ts, key, 2 );
   idx[3] = hash( ts, key, 3 );

   Slot entry;
   for( int i = 0; i < 4; ++i )
   {
      entry = ts->table[idx[i] % ts->capacity];
      Key k = slotKey( entry );
      if( k == key )
         return;
      if( k == NULL_KEY )
         break;
   }

   // Should never fail except for invalid keys
   printf( "Query for %u failed\n", key );
}
}

#define ERROR_CHECK(x,eType,eSuccess,eStrFn)\
   do\
   {\
      eType err = x;\
      if( err != eSuccess )\
      {\
         printf( "Error (%d:%s) at %s:%d\n", err, eStrFn, __FILE__, __LINE__ );\
         return;\
      }\
   } while( false )
#define CUDA_CALL(x) ERROR_CHECK(x,hipError_t,hipSuccess,hipGetErrorString(err))

// Copy host data into device side arrays
void copyData( bool cuckoo, int N, Key* keys, Value* values, uint32_t* params, int numParams )
{
   assert( numParams >= 8 );

   _inputSize = N;
   int capacity = N / LOAD_FACTOR;
   size_t keySize = N * sizeof(Key);
   size_t valSize = N * sizeof(Value);
   size_t paramSize = numParams * sizeof(uint32_t);
   size_t tableSize = capacity * sizeof(Slot);
   
   CUDA_CALL(hipMalloc( &_tableState, sizeof(TableState) ));
   CUDA_CALL(hipMalloc( &_keys, keySize ));
   CUDA_CALL(hipMalloc( &_values, valSize ));
   CUDA_CALL(hipMalloc( &_params, paramSize ));
   CUDA_CALL(hipMalloc( &_table, tableSize ));
   CUDA_CALL(hipMemcpy( _keys, keys, keySize, hipMemcpyHostToDevice ));
   CUDA_CALL(hipMemcpy( _values, values, valSize, hipMemcpyHostToDevice ));
   CUDA_CALL(hipMemcpy( _params, params, paramSize, hipMemcpyHostToDevice ));
   CUDA_CALL(hipMemset( _table, 0, tableSize ));

   initTable <<<1,1>>> ( cuckoo, _tableState, capacity, _table, _params, numParams, _inputSize );

   // Synchronize to get accurate timing
   CUDA_CALL(hipDeviceSynchronize());
}

// Allocate one thread to insert each input item
void constructTable( bool cuckoo )
{
   // Calculate reasonable grid/block dimensions
   const int maxBlockSize = 64;
   _grid  = dim3( 1, 1, 1 );
   _block = dim3( 1, 1, 1 );
   if( _inputSize < maxBlockSize )
   {
      _block.x = _inputSize;
   }
   else
   {
      _block.x = maxBlockSize;
      _grid.x  = _inputSize / maxBlockSize;
      assert( _inputSize % maxBlockSize == 0 );
   }

   printf( "Launching %d x %d threads\n", _grid.x, _block.x );

   if( cuckoo )
      cuckooInsert <<<_grid,_block>>> ( _tableState, _keys, _values );
   else
      quadInsert <<<_grid,_block>>> ( _tableState, _keys, _values );

   CUDA_CALL(hipDeviceSynchronize());
}

void queryTable( bool cuckoo, int times )
{
   if( cuckoo )
   {
      for( int i = 0; i < times; ++i )
         cuckooQuery <<<_grid,_block>>> ( _tableState, _keys, _values );
   }
   else
   {
      for( int i = 0; i < times; ++i )
         quadQuery <<<_grid,_block>>> ( _tableState, _keys, _values );
   }

   CUDA_CALL(hipDeviceSynchronize());
}

void tearDown()
{
   CUDA_CALL(hipFree( _keys ));
   CUDA_CALL(hipFree( _values ));
   CUDA_CALL(hipFree( _params ));
   CUDA_CALL(hipFree( _table ));
   CUDA_CALL(hipFree( _tableState ));
}

